#include "hip/hip_runtime.h"
#include "dataloader.cuh"

float* allocatePinnedMemory(size_t size) {
    float* ptr;
    hipHostMalloc(&ptr, size * sizeof(float));
    return ptr;
}

void freePinnedMemory(float* ptr) {
    hipHostFree(ptr);
}

DataLoader::DataLoader(std::vector<std::vector<float>>& data, std::vector<int>& labels, size_t batchsize, size_t cropping_size, bool shuffle, size_t num_workers, bool pin_memory, bool drop_last) {
    this->batchsize = batchsize;
    this->num_workers = num_workers;
    this->pin_memory = pin_memory;
    this->drop_last = drop_last;
    this->shuffle = shuffle;

    this->idx = 0;
    this->cropping_size = cropping_size; // cropping length, default: 21950
    // this->channel = channel;

    this->data = data;
    this->labels = labels;

    if(shuffle) {
        std::random_device rd;
        std::mt19937 g(rd());

        std::shuffle(data.begin(), data.end(), g);
    }
}

std::vector<float> DataLoader::crop(std::vector<float>& points) {
    std::vector<float> cropped_points(3 * cropping_size, 0.0f);
    size_t size = std::min(points.size(), this->cropping_size);
    std::copy(points.begin(), points.begin() + size, cropped_points.begin());
    return cropped_points;
}


Tensor* DataLoader::getBatchedData(std::vector<int>& labels) {
    DimVector shape;

    if (this->pin_memory) {
        // TODO: Implement pin_memory handling
        return nullptr;
    } else {
        std::vector<float> batch;

        if (this->idx < data.size()) {
            if (this->idx + batchsize > data.size() && !drop_last) {
                shape = {this->data.size() - this->idx, cropping_size, 3};
                batch.reserve((this->data.size() - this->idx) * cropping_size * 3);

                for (; this->idx < this->data.size(); this->idx++) {
                    std::vector<float> point = batchsize > 1 ? crop(this->data[this->idx]) : this->data[this->idx];
                        
                    batch.insert(batch.end(), point.begin(), point.end());
                }
                labels.assign(this->labels.begin() + this->idx, this->labels.end());
                this->idx = data.size();
            } else {
                if(batchsize > 1) {
                    shape = {batchsize, cropping_size, 3};
                    batch.reserve(batchsize * cropping_size * 3);
                } else {
                    shape = {1, this->data[this->idx].size()/3, 3};
                    batch.reserve(this->data[this->idx].size());
                }
                if (this->idx + batchsize > this->labels.size()) {
                    throw std::out_of_range("Index out of range for labels");
                }
                labels.assign(this->labels.begin() + this->idx, this->labels.begin() + this->idx + batchsize);

                for (int i = 0; i < batchsize; i++) {
                    if (this->idx >= this->data.size()) {
                        throw std::out_of_range("Index out of range for data");
                    }
                    std::vector<float> point = batchsize > 1 ? crop(this->data[this->idx]) : this->data[this->idx];
                    batch.insert(batch.end(), point.begin(), point.end());
                    this->idx++;
                }
            }
        } else {
            throw std::runtime_error("No more data to load");
        }

        Tensor* batch_input = new Tensor(shape);
        batch_input->fromVec(batch);

        return batch_input;
    }
}


size_t DataLoader::getDataNum(){
    return (size_t)this->labels.size();
}

size_t DataLoader::getBatchNum() {
    return (size_t)(!drop_last ? (this->getDataNum()-1) / batchsize + 1:this->getDataNum() / batchsize);
}