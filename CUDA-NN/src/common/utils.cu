#include "utils.cuh"

void setGPU(const int GPU_idx){
    int count = 0;
    CHECK(hipGetDeviceCount(&count));
    CHECK(hipSetDevice(GPU_idx));
}


float* loadWeightsFromTxt(const char* filename, std::vector<size_t> shape) {
    FILE* file = fopen(filename, "r");
    if (file == NULL) {
        perror("Error opening file");
        return NULL;
    }

    int dim = shape.size();
    size_t n_data = 1;
    for(size_t s: shape) {
        n_data *= s;
    }
    int nBytes = n_data * sizeof(float);

    float *matrix = (float *) malloc(nBytes);

    if (matrix == NULL) {
        perror("Error allocating memory");
        fclose(file);
        return NULL;
    }


    if(dim == 1) {
        for (int i=0; i<shape[0]; i++) {
            if(fscanf(file, "%f", &matrix[i]) != 1) {
                free(matrix);
                fclose(file);
                return NULL;
            }
        }
    }else if(dim == 2) {
        for (int i=0; i<shape[0]; i++) {
            for (int j=0; j<shape[1]; j++) {
                if(fscanf(file, "%f", &matrix[i*shape[1] + j]) != 1) {
                    free(matrix);
                    fclose(file);
                    return NULL;
                }
            }
        }
    } else if(dim == 3) {
        for (int i=0; i<shape[0]; i++) {
            for (int j=0; j<shape[1]; j++) {
                for(int k=0; k<shape[2]; k++) {
                    if(fscanf(file, "%f", &matrix[i*shape[1]*shape[2] + j*shape[2] + k]) != 1) {
                        free(matrix);
                        fclose(file);
                        return NULL;
                    }
                }
            }
        }
    } else {
        ERROR("Dimension Error!");
    }

    fclose(file);
    return matrix;
}

void printM(float* weight, const std::vector<size_t> shape) {
    printShape(shape);
    int dim = shape.size();
    if (weight == NULL) {
        printf("print_M: Matrix is NULL\n");
        return;
    }
    if (dim == 1) {
        int m = shape[0];
        for(int i=0; i<m; i++) {
            printf("%.3f ", weight[i]);
        }
        printf("\n");
    } else if (dim == 2) {
        int m = shape[0], n = shape[1];
        for (int i=0; i<m; i++) {
            for (int j=0; j<n; j++) {
                printf("%.3f ", weight[i*n + j]);
            }
            printf("\n");
        }
        printf("\n");
    } else if(dim == 3) {
        int bz = shape[0], m = shape[1], n = shape[2];
        for(int b = 0; b < bz; b++) {
            printf("--------------batch %d---------------\n", b+1);
            for (int i=0; i<m; i++) {
                printf("[");
                for (int j=0; j<n; j++) {
                    printf("%.3f ", weight[b * m * n + i * n + j]);
                }
                printf("]\n");
            }
        }
    } else {
        printf("print_M: dim > 3 not implemented!");
        exit(0);
    }
}

void printShape(std::vector<size_t> shape) {
    printf("shape: (");
    for(int i=0; i<shape.size() - 1; i++) {
        printf("%ld ", shape[i]);
    }
    printf("%ld)\n", shape[shape.size()-1]);
}


float randomFloat(float a, float b) {
    return a + static_cast <float> (std::rand()) /( static_cast <float> (RAND_MAX/(b-a)));
}

int randomInt(int a, int b) {
    return a + std::rand()%(b-a);
}