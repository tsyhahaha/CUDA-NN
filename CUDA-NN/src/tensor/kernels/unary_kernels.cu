#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "configure.cuh"


__global__
void kMaxLastDim3D(float* d_data, float* d_out, size_t N, size_t C, size_t L
) {
    // It'll be faster if blocksize is the factor of L.
    int x = blockIdx.x;
    int y = blockIdx.y;
    int tid = threadIdx.x;

    // if(x >= C || y >= N) return;

    __shared__ float sd_M[BLOCK_SIZE1D];
    float cur_max = 0.0f;

    int iter = (L-1)/BLOCK_SIZE1D + 1;
    for(int i=0; i<iter; i++) {
        if (i*BLOCK_SIZE1D + tid < L) {
            sd_M[tid] = d_data[(y*C*L + x*L) + i*BLOCK_SIZE1D + tid];
        }
        __syncthreads();

        // reduce max and save to `cur_max`
        for(int stride=blockDim.x/2; stride>0; stride>>=1) {
            if(tid < stride && tid + stride + i*BLOCK_SIZE1D < L) {
                sd_M[tid] = sd_M[tid] > sd_M[tid + stride]? sd_M[tid] : sd_M[tid+stride];
            }
            __syncthreads();
        }
        cur_max = cur_max >= sd_M[0] ? cur_max : sd_M[0];
    }


    if (tid==0 && y < N && x < C)
        // printf("d_out[%d][%d] = %f\n",y, x, cur_max);
        d_out[y * C + x] = cur_max;
}

__global__
void kTransposeLast3D(float* d_data, float* d_out, size_t N, size_t m, size_t n
){
    __shared__ float sd_M[BLOCK_SIZE2D][BLOCK_SIZE2D];

    for(int b=0; b<N; b++) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= n || y >= m) {
            return;
        }
        // printf("d_data[%d][%d][%d] = %f\n", b, y, x, d_data[b*m*n + y * n + x]);
        sd_M[threadIdx.y][threadIdx.x] = d_data[b*m*n + y * n + x];

        __syncthreads();

        d_out[b*m*n + x * m + y] = sd_M[threadIdx.y][threadIdx.x];
    }
}

__global__
void kScale(float *d_data, float factor, float offset, size_t N) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    d_data[tid] = d_data[tid] * factor + offset;
}

__global__
void kSum(float *d_M, float *d_out, int N) {
    __shared__ float sd_M[BLOCK_SIZE1D];

    int tid = threadIdx.x;
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    int block_num = gridDim.x;

    ///////////////////////////////////////////////
    // the first loop: read from d_M
    if (gidx < N) sd_M[tid] = d_M[gidx];
    else sd_M[tid] = 0;
    __syncthreads();

    for(int stride=blockDim.x/2; stride>0; stride>>=1) {
        if(tid < stride && tid + stride < N) {
            sd_M[tid] += sd_M[tid + stride];
        }
        __syncthreads();
    }
    if(tid == 0) d_out[blockIdx.x] = sd_M[0];
    
    N = block_num;
    block_num = (block_num-1)/blockDim.x + 1;
    gidx = (gidx-1) / blockDim.x + 1;
    //////////////////////////////////////////////////

    while(N >= blockDim.x) {
        if (gidx < N) 
            sd_M[tid] = d_out[gidx];
        else 
            sd_M[tid] = 0;
        __syncthreads();

        // reduce the shared memory
        for(int stride=blockDim.x/2; stride>0; stride>>=1) {
            if(tid < stride && tid + stride < N) {
                sd_M[tid] += sd_M[tid + stride];
            }
            __syncthreads();
        }

        if(tid == 0) d_out[blockIdx.x] = sd_M[0];

        N = block_num;
        block_num = (N-1)/blockDim.x + 1;
        gidx = (gidx-1) / blockDim.x + 1;
    }


    // N < BLOCK_SIZE
    if(tid < N) sd_M[tid] = d_out[tid];
    else        sd_M[tid] = 0;

    for(int stride=blockDim.x/2; stride>0; stride>>=1) {
        if(tid < stride) {
            sd_M[tid] += sd_M[tid + stride];
        }
        __syncthreads();
    }

    if (tid==0) d_out[0] = sd_M[0];
}


__global__ 
void kTranspose(float* d_M, float* d_out, int m, int n) {
    __shared__ float sd_M[BLOCK_SIZE2D][BLOCK_SIZE2D];
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= n || y >= m) {
        return;
    }

    sd_M[threadIdx.y][threadIdx.x] = d_M[y * n + x];

    __syncthreads();

    d_out[x * m + y] = sd_M[threadIdx.y][threadIdx.x];
}









