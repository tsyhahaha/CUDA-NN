#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "stdio.h"

__global__
void kAdd_l3(
    float *d_A, float *d_B, float *d_out, int M, int N, float f1, float f2
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= M || col >= N) return;

    d_out[row * N + col] = f1 * d_A[row * N + col] + f2 * d_B[row * N + col];
}

__global__
void kMatmul_l3(
    float *d_A, float *d_B, float *d_out, int M, int N, int K
) {
    float cVal = 0.0f;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float ds_A[BLOCK_SIZE2D][TILE_SIZE];
    __shared__ float ds_B[TILE_SIZE][BLOCK_SIZE2D];

    int phase = (N - 1) / TILE_SIZE + 1;
    for(int p=0; p<phase;p++) {
        if (row < M && p*TILE_SIZE + threadIdx.x < N && threadIdx.x < TILE_SIZE) 
        ds_A[threadIdx.y][threadIdx.x] = d_A[row*N + p*TILE_SIZE + threadIdx.x];
        if(p*TILE_SIZE + threadIdx.y < N && col < K && threadIdx.y < TILE_SIZE)
        ds_B[threadIdx.y][threadIdx.x] = d_B[(p*TILE_SIZE + threadIdx.y)*K + col];

        __syncthreads();
        for (int i=0; i<TILE_SIZE; i++) {
            // constant: ds_A's x , ds_B's y
            cVal += ds_A[threadIdx.y][i] * ds_B[i][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < M && col < K)
        d_out[row*K + col] = cVal;
}

__global__
void kMatmulTransposed_l3(
    float *d_A, float *d_B, float *d_out, int M, int N, int K
) {
    float cVal = 0.0f;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= M || col >= K) return;

    // TILE_SIZE == BLOCK_SIZE2D
    __shared__ float ds_A[BLOCK_SIZE2D][TILE_SIZE];
    __shared__ float ds_B[BLOCK_SIZE2D][TILE_SIZE];

    int phase = (N - 1) / TILE_SIZE + 1;
    for(int p=0; p<phase;p++) {
        if (row < M && p*TILE_SIZE + threadIdx.x < N && threadIdx.x < TILE_SIZE) {
            // ds_A[ty][tx] = d_A[row][p*TILE_SIZE + tx]
            ds_A[threadIdx.y][threadIdx.x] = d_A[row*N + p*TILE_SIZE + threadIdx.x];
        }
            
        if (col < K && p*TILE_SIZE + threadIdx.y < N && threadIdx.y < TILE_SIZE) {
            // ds_B[tx][ty] = d_B[col][p*TILE_SIZE + ty]
            ds_B[threadIdx.x][threadIdx.y] = d_B[col*N + p*TILE_SIZE + threadIdx.y]; 
        }

        __syncthreads();
        for (int i=0; i<TILE_SIZE; i++) {
            // constant: ds_A's x , ds_B's x
            cVal += ds_A[threadIdx.y][i] * ds_B[threadIdx.x][i];
        }
        __syncthreads();
    }

    printf("M=%d N=%d K=%d, d_out[%d][%d] = %f\n", M, N, K, row, col, cVal);
    d_out[row*K + col] = cVal;
}

__global__
void kMatmulStride_l3(
    float*d_A, float* d_B, float*d_out, int M, int N 
) {
    // A (B x M x N) @ B (B/1 x N x K) = C (B x M x K)
          
}