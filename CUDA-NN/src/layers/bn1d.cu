#include "hip/hip_runtime.h"
#include "bn1d.cuh"

/* [(N x C) - (C)] / sqrt(C + eps) * (C) + (C) */
__global__ 
void kBn1d_l2(
    float* d_data, float* d_out, 
    float* weights, float* bias, float* mean, float* var, 
    float eps, int N, int C, bool relu
) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row >= N || col >= C) return;

    float scaling = rsqrtf(var[col] + eps); // 1 / sqrt(C + eps)
    float norm = (d_data[row * C + col] - mean[col]) * scaling;

    float cVal = norm * weights[col] + bias[col];
    d_out[row * C + col] = relu ? fmaxf(cVal, 0.0) : cVal;
}

__global__ 
void kBn1d_cache_l2(
    float* d_data, float* d_out, 
    float* weights, float* bias, float* mean, float* var, 
    float* x_hat, float* x_minus_mu, float* sqrt_var_inv,
    float eps, int N, int C, bool relu
) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row >= N || col >= C) return;
    int offset2d = row*C + col;

    float scaling = rsqrtf(var[col] + eps); // 1 / sqrt(C + eps)
    float x_m = d_data[offset2d] - mean[col];
    float norm = x_m * scaling;

    float cVal = norm * weights[col] + bias[col];
    d_out[offset2d] = relu ? fmaxf(cVal, 0.0) : cVal;

    // cache
    x_hat[offset2d] = norm;
    x_minus_mu[offset2d] = x_m;
    if(threadIdx.y == 0)
        sqrt_var_inv[col] = scaling;
}


/* [(N x C x L) - (C)] / sqrt(C + eps) * (C) + (C) */
__global__ 
void kBn1d_l3(
    float* d_data, float* d_out, 
    float* weights, float* bias, float* mean, float* var,
    float eps, int N, int C, int L, bool relu
) {
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if(z >= N || y >= C || x >= L) return;

    float scaling = rsqrtf(var[y] + eps); // 1 / sqrt(C + eps)
    float norm = (d_data[z*C*L + y*L + x] - mean[y]) * scaling;

    float cVal = norm * weights[y] + bias[y];
    d_out[z*C*L + y*L + x] = relu ? fmaxf(cVal, 0.0) : cVal;
}
/* [(N x C x L) - (C)] / sqrt(C + eps) * (C) + (C) */
__global__ 
void kBn1d_cache_l3(
    float* d_data, float* d_out, 
    float* weights, float* bias, float* mean, float* var, 
    float* x_hat, float* x_minus_mu, float* sqrt_var_inv,
    float eps, int N, int C, int L, bool relu
) {
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if(z >= N || y >= C || x >= L) return;

    int offset3d = z*C*L + y*L + x;

    float scaling = rsqrtf(var[y] + eps); // 1 / sqrt(C + eps)
    
    float x_m = d_data[offset3d] - mean[y];
    float norm = x_m * scaling;
    

    float cVal = norm * weights[y] + bias[y];
    if(cVal > 1000) {
        printf("x_n = %f, %f = 1/(%f + %f), weight = %f, bias = %f\n", x_m, scaling, var[y], eps, weights[y], bias[y]);
    }
    d_out[offset3d] = relu ? fmaxf(cVal, 0.0) : cVal;

    // cache
    x_hat[offset3d] = norm;
    x_minus_mu[offset3d] = x_m;
    if(threadIdx.x == 0 && threadIdx.z == 0)
        sqrt_var_inv[y] = scaling;
}

__global__
void kBackprop_to_mean_and_var_l2(
    float* d_mean, float* d_var, float* d_x_hat, float* sqrt_var_inv, float* x_minus_mu, int N, int C, int stride
) {
    int row = threadIdx.y;  // single row of width: BATCH_BASE
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    
    // float d_m = 0.0f, d_v = 0.0f;
    __shared__ float ds_var[BLOCK_SIZE1D][BATCH_BASE];
    __shared__ float ds_mean[BLOCK_SIZE1D][BATCH_BASE];

    if(col >= C) return;

    int iter = (N-1)/BATCH_BASE + 1;
    for(int i=0; i<iter; i++) {
        int real_row = (row + i*BATCH_BASE);
        int idx2d = real_row * C + col;
        int idx1d = col / stride;
        int tid = threadIdx.y;

        if(real_row < N) {
            float tmp = - d_x_hat[idx2d] * sqrt_var_inv[idx1d];
            ds_mean[threadIdx.x][threadIdx.y] = tmp;
            ds_var[threadIdx.x][threadIdx.y] = 0.5 * tmp * x_minus_mu[idx2d] * powf(sqrt_var_inv[idx1d], 2);
            __syncthreads();
        }
            

        for(int stride=BATCH_BASE/2; stride>0; stride>>=1) {
            if(tid < stride && tid + stride + i*BATCH_BASE < N) {
                ds_mean[threadIdx.x][tid] = ds_mean[threadIdx.x][tid] + ds_mean[threadIdx.x][tid + stride];
                ds_var[threadIdx.x][tid] = ds_var[threadIdx.x][tid] + ds_var[threadIdx.x][tid + stride];
            }
            __syncthreads();
        }

        if(threadIdx.y == 0 && col < C && real_row < N) {
            d_mean[idx1d] = ds_mean[threadIdx.x][0];
            d_var[idx1d] = ds_var[threadIdx.x][0];
        }
    }
}

__global__
void kBn1d_back_l2(
    float* d_in, 
    float* d_x_hat, float* d_var, float* d_mean,
    float* sqrt_var_inv, float* x_minus_mu, int N, int C
) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    if(col >= C || row >= N) return;

    int idx2d = row * C + col;
    float f1 = sqrt_var_inv[col];
    float f2 = 2 * x_minus_mu[idx2d] / N;

    d_in[idx2d] = d_x_hat[idx2d] * f1 + d_var[col] * f2 + d_mean[col] / N;
}

__global__
void kBn1d_back_l3(
    float* d_in, 
    float* d_x_hat, float* d_var, float* d_mean,
    float* sqrt_var_inv, float* x_minus_mu, int N, int C, int L
) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int batch = threadIdx.z + blockDim.z * blockIdx.z;

    if(col >= L || row >= C || batch >= N) return;

    int idx3d = batch * C * L + row * L + col;

    float f1 = sqrt_var_inv[row];
    float f2 = 2 * x_minus_mu[idx3d] / N;

    d_in[idx3d] = d_x_hat[idx3d] * f1 + d_var[row] * f2 + d_mean[row] / N;
}

__global__
void kBackprop_to_mean_and_var_l3(
    float* d_mean, float* d_var, float* d_x_hat, float* sqrt_var_inv, float* x_minus_mu, int N, int C, int L
) {
    
    int row = threadIdx.y;  // single row of width: BATCH_BASE
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    
    // float d_m = 0.0f, d_v = 0.0f;
    __shared__ float ds_var[BLOCK_SIZE1D][BATCH_BASE];
    __shared__ float ds_mean[BLOCK_SIZE1D][BATCH_BASE];

    if(col >= C) return;

    int iter = (N-1)/BATCH_BASE + 1;

    for(int i=0; i<iter; i++) {
        int real_row = (row + i*BATCH_BASE);
        int idx2d = real_row * C + col;
        int idx1d = col;
        int tid = threadIdx.y;

        float tmp = - d_x_hat[idx2d] * sqrt_var_inv[idx1d];
        
        ds_mean[threadIdx.x][threadIdx.y] = tmp;
        ds_var[threadIdx.x][threadIdx.y] = 0.5 * tmp * x_minus_mu[idx2d] * powf(sqrt_var_inv[idx1d], 2);

        __syncthreads();

        for(int stride=BATCH_BASE/2; stride>0; stride>>=1) {
            if(tid < stride && tid + stride + i*BATCH_BASE < N) {
                ds_mean[threadIdx.x][tid] = ds_mean[threadIdx.x][tid] + ds_mean[threadIdx.x][tid + stride];
                ds_var[threadIdx.x][tid] = ds_var[threadIdx.x][tid] + ds_var[threadIdx.x][tid + stride];
            }
            __syncthreads();
        }

        if(threadIdx.y == 0 && col < C) {
            d_mean[idx1d] = ds_mean[threadIdx.x][0];
            d_var[idx1d] = ds_var[threadIdx.x][0];
        }
    }
}

__global__
void kBackprop_to_weights_and_bias_l3(
    float* d_mean, float* d_var, float* d_x_hat, float* sqrt_var_inv, float* x_minus_mu, int N, int C, int L
) {
    
}

BatchNorm1d::BatchNorm1d(std::string prefix, size_t num_features, bool relu, float eps, float monmentum, bool affine, bool track_running_stats) {
    this->num_features = num_features;
    this->relu = relu;
    this->eps = eps;
    this->momentum = monmentum;
    this->affine = affine;
    this->track_running_stats = track_running_stats;

    this->prefix = prefix;

    if(affine) {
        this->weights = new Tensor({num_features}, NONE);
        this->bias = new Tensor({num_features}, NONE);
    } else {
        // the output is just centralization
        this->weights = nullptr;
        this->bias = nullptr;
    }

    if(this->track_running_stats) {
        this->running_mean = new Tensor({num_features}, NONE);
        this->running_var = new Tensor({num_features}, NONE);
        if(this->is_training) {
            this->x_minus_mu = new Tensor({num_features}, NONE);
            this->sqrt_var_inv = new Tensor({num_features}, NONE);
        }
    } else {
        // use batch statistics(bias estimate)
        this->running_mean = nullptr;
        this->running_var = nullptr;
    }
}

BatchNorm1d::BatchNorm1d(size_t num_features, bool relu, float eps, float monmentum, bool affine, bool track_running_stats) {
    this->num_features = num_features;
    this->relu = relu;
    this->eps = eps;
    this->momentum = monmentum;
    this->affine = affine;
    this->track_running_stats = track_running_stats;

    if(affine) {
        this->weights = new Tensor({num_features}, NONE);
        this->bias = new Tensor({num_features}, NONE);
    } else {
        // the output is just centralization
        this->weights = nullptr;
        this->bias = nullptr;
    }

    if(this->track_running_stats) {
        this->running_mean = new Tensor({num_features}, NONE);
        this->running_var = new Tensor({num_features}, NONE);
        if(this->is_training) {
            this->x_minus_mu = new Tensor({num_features}, NONE);
            this->sqrt_var_inv = new Tensor({num_features}, NONE);
        }
    } else {
        // use batch statistics(bias estimate)
        this->running_mean = nullptr;
        this->running_var = nullptr;
    }
}

BatchNorm1d::~BatchNorm1d() {
    if(this->track_running_stats) {
        delete running_mean;
        delete running_var;
    }
    if(x_minus_mu!= nullptr) delete x_minus_mu;
    if(sqrt_var_inv != nullptr) delete sqrt_var_inv;
    if(x_hat!= nullptr) delete x_hat;
}

BatchNorm1d* BatchNorm1d::train() {
    BaseLayer::train();
    this->is_training = true;
    size_t bz = Configurer::batch_size;
    size_t l = Configurer::cropping_size;
    if(!d_in) {
        d_in = new Tensor({bz, num_features, l});
    }
    return this;
}

void BatchNorm1d::load_weights() {
    this->weights->fromVec(Configurer::getWeights(this->prefix + "weight"));
    this->bias->fromVec(Configurer::getWeights(this->prefix + "bias"));
    this->running_mean->fromVec(Configurer::getWeights(this->prefix + "running_mean"));
    this->running_var->fromVec(Configurer::getWeights(this->prefix + "running_var"));
}

void BatchNorm1d::init_weights() {
    if(affine) {
        DEBUG_PRINT("BatchNorm1d init weights: ONES\n");
        DEBUG_PRINT("BatchNorm1d init bias: ZERO\n");
        this->weights->initialize(ONES);
        this->bias->initialize(ZERO);
    }

    if(this->track_running_stats) {
        DEBUG_PRINT("BatchNorm1d init mean: ZERO\n");
        DEBUG_PRINT("BatchNorm1d init var: ONES\n");
        this->running_mean->initialize(ZERO);
        this->running_var->initialize(ONES);
    }
}

void BatchNorm1d::load_weights(std::vector<float>& params, const std::string& target) {
    size_t n_data = params.size();
    float* h_data = params.data();
    if(target=="weights") {
        assert(n_data == this->weights->getSize());
        this->weights->load(h_data, n_data);
    } else if(target == "bias") {
        assert(n_data == this->bias->getSize());
        this->bias->load(h_data, n_data);
    } else if(target == "mean") {
        assert(this->running_mean->getSize() == n_data);
        this->running_mean->load(h_data, n_data);
    } else if(target == "var") {
        assert(this->running_var->getSize() == n_data);
        this->running_var->load(h_data, n_data);
    } else {
        ERROR("Load weights %s error!\n", target.c_str());
    }
}

void BatchNorm1d::prepare_backward() {
    // cache for backward
    DimVector shape = input->getShape();
    DimVector point_shape = shape;
    point_shape.erase(point_shape.begin());
    if(!x_hat) {
        x_hat = new Tensor(shape);
    } x_hat->reset(shape);

    if(!x_minus_mu){
        x_minus_mu = new Tensor(shape);
    } x_minus_mu->reset(shape);

    if(!sqrt_var_inv) {
        sqrt_var_inv = new Tensor({num_features});
    } sqrt_var_inv->reset({num_features});

    if(!d_var) {
        d_var = new Tensor({num_features});
    } d_var->reset({num_features});
    if(!d_mean) {
        d_mean = new Tensor({num_features});
    } d_mean->reset({num_features});
}

Tensor* BatchNorm1d::forward(Tensor* data) {
    DEBUG_PRINT("[BatchNorm1d] %sforward\n", this->prefix.c_str());
    if(!this->track_running_stats){
        ERROR("not implemented!");
    }
    DimVector shape_o = data->getShape();

    if(this->output == nullptr) {
        this->output = new Tensor(shape_o);
    } this->output->reset(shape_o);

    if(this->is_training) {
        this->input = data;

        input->mean(mean_cache, 0);
        if(mean_cache->getDim() > 1) {
            mean_cache->mean_(1, true);
        }

        // unroll var function
        input->sub(var_cache, mean_cache);
        var_cache->square_();
        var_cache->mean_(0);
        if(var_cache->getDim() > 1) {
            var_cache->mean_(1);
        }

        // float* mean_tmp = mean_cache->toHost();
        // float* var_tmp = var_cache->toHost();
        // float* input_tmp = input->toHost();
        // printM(input_tmp, input->getShape());
        // printM(mean_tmp, mean_cache->getShape());
        // printM(var_tmp, var_cache->getShape());
        // free(mean_tmp); free(var_tmp); free(input_tmp);

        prepare_backward();
    }


    size_t dim = data->getDim();
    if(dim == 2) {

        DimVector shape = data->getShape();
        int N = shape[0], C = shape[1];

        dim3 block(BLOCK_SIZE1D, BATCH_BASE);
        dim3 grid((C-1)/BLOCK_SIZE1D + 1, (N-1)/BATCH_BASE+1); 

        if(!is_training) {
            kBn1d_l2<<<grid, block>>>(data->getData(), output->getData(), weights->getData(), bias->getData(), running_mean->getData(), running_var->getData(), eps, N, C, this->relu); CHECK_KERNEL();
        } else {
            kBn1d_cache_l2<<<grid, block>>>(data->getData(), output->getData(), weights->getData(), bias->getData(), mean_cache->getData(), var_cache->getData(), x_hat->getData(), x_minus_mu->getData(), sqrt_var_inv->getData(), eps, N, C, this->relu); CHECK_KERNEL();
        }

    } else if(dim == 3) {
        DimVector shape = data->getShape();
        int N = shape[0], C = shape[1], L = shape[2];

        dim3 block(BLOCK_SIZE3D, BLOCK_SIZE3D, BATCH_BASE);
        dim3 grid((L-1)/BLOCK_SIZE3D + 1, (C-1)/BLOCK_SIZE3D+1, (N-1)/BATCH_BASE+1); 
        if(!is_training) {
            kBn1d_l3<<<grid, block>>>(data->getData(), this->output->getData(),  weights->getData(), bias->getData(), running_mean->getData(), running_var->getData(), eps, N, C, L, this->relu); CHECK_KERNEL();
        } else {
            kBn1d_cache_l3<<<grid, block>>>(data->getData(), output->getData(), weights->getData(), bias->getData(), mean_cache->getData(), var_cache->getData(), x_hat->getData(), x_minus_mu->getData(), sqrt_var_inv->getData(), eps, N, C, L, this->relu); CHECK_KERNEL();
        }
    } else {
        ERROR("Dimension not allowed!");
    }
    if(is_training) {
        running_mean->add_(mean_cache->squeeze(), (1 - momentum), momentum);
        running_var->add_(var_cache->squeeze(), (1 - momentum), momentum);
    }

    return this->output;
}


Tensor* BatchNorm1d::backward(Tensor* gradients){
    DEBUG_PRINT("[BatchNorm1d] %sbackward\n", this->prefix.c_str());
    if(this->relu) {
        gradients->mask(output);
    }

    // gradients(N, C, L)
    size_t size = input->getDim();
    DimVector shape = input->getShape();

    // (N, C)/(N, C, L) * (N, C)/(N, C, L) -> sum ->(C)
    gradients->dot(d_weights, x_hat->squeeze());

    d_weights->sumToDim_(1);
    gradients->sumToDim(d_bias, 1);

    // accumulate grads
    weights->acc_grads(d_weights);
    bias->acc_grads(d_bias);

    // backward kernel
    if(size == 2) {
        size_t N = shape[0], C = shape[1];
        this->d_in->reset({N, C});
        gradients->dot(d_x_hat, weights); // (N,C) * (C)
        dim3 block(BLOCK_SIZE1D, BATCH_BASE);
        dim3 grid((C-1)/BLOCK_SIZE1D+1, 1);

        kBackprop_to_mean_and_var_l2<<<grid, block>>>(d_mean->getData(), d_var->getData(), d_x_hat->getData(), sqrt_var_inv->getData(), x_minus_mu->getData(), N, C, 1);CHECK_KERNEL();

        dim3 block_sub(BLOCK_SIZE1D, BATCH_BASE);
        dim3 grid_sub((C-1)/BLOCK_SIZE1D+1, (N-1)/BATCH_BASE+1);

        kBn1d_back_l2<<<grid_sub, block_sub>>>(d_in->getData(), d_x_hat->getData(), d_var->getData(), d_mean->getData(), sqrt_var_inv->getData(), x_minus_mu->getData(), N, C);CHECK_KERNEL();
    } else if(size==3) {
        size_t N = shape[0], C = shape[1], L = shape[2];
        this->d_in->reset({N, C, L});
        // DEBUG_PRINT("(%d, %d, %d)\n", N, C, L);
        weights->unsqueeze(-1);
        gradients->dot(d_x_hat, weights); //(N,C,L)*(C,1)
        weights->squeeze(-1);

        dim3 block(BLOCK_SIZE1D, BATCH_BASE);
        dim3 grid((C*L-1)/BLOCK_SIZE1D+1, 1);

        // printShape(d_mean->getShape());         // (C)
        // printShape(d_var->getShape());          // (C)
        // printShape(sqrt_var_inv->getShape());   // (C)
        // printShape(d_x_hat->getShape());        // (N, C, L)
        // printShape(x_minus_mu->getShape());     // (N, C, L)

        kBackprop_to_mean_and_var_l2<<<grid, block>>>(d_mean->getData(), d_var->getData(), d_x_hat->getData(), sqrt_var_inv->getData(), x_minus_mu->getData(), N, C*L, L);CHECK_KERNEL();

        dim3 block_sub(BLOCK_SIZE2D, BLOCK_SIZE2D, BATCH_BASE);
        dim3 grid_sub((L-1)/BLOCK_SIZE2D+1, (C-1)/BLOCK_SIZE1D+1, (N-1)/BATCH_BASE+1);
        kBn1d_back_l3<<<grid_sub, block_sub>>>(d_in->getData(), d_x_hat->getData(), d_var->getData(), d_mean->getData(), sqrt_var_inv->getData(), x_minus_mu->getData(), N, C, L);CHECK_KERNEL();
    } else {
        ERROR("Size not matched!\n");
    }

    return d_in;
}
