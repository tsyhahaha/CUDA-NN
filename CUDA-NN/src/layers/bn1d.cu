#include "hip/hip_runtime.h"
#include "bn1d.cuh"
#include "kernels.cuh"

/* [(N x C) - (C)] / sqrt(C + eps) * (C) + (C) */
__global__ 
void kBn1d_l2(float* d_data, float* d_out, float* weights, float* bias, 
    float* mean, float* var, float eps, int N, int C, bool relu
) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row >= N || col >= C) return;

    float scaling = rsqrtf(var[col] + eps); // 1 / sqrt(C + eps)
    float norm = (d_data[row * C + col] - mean[col]) * scaling;

    float cVal = norm * weights[col] + bias[col];
    d_out[row * C + col] = relu ? fmaxf(cVal, 0.0) : cVal;
}


/* [(N x C x L) - (C)] / sqrt(C + eps) * (C) + (C) */
__global__ 
void kBn1d_l3(float* d_data, float* d_out, float* weights, float* bias, 
    float* mean, float* var, float eps, int N, int C, int L, bool relu
) {
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if(z >= N || y >= C || x >= L) return;

    float scaling = rsqrtf(var[y] + eps); // 1 / sqrt(C + eps)
    float norm = (d_data[z*C*L + y*L + x] - mean[y]) * scaling;

    float cVal = norm * weights[y] + bias[y];
    d_out[z*C*L + y*L + x] = relu ? fmaxf(cVal, 0.0) : cVal;
}

BatchNorm1d::BatchNorm1d(std::string prefix, size_t num_features, bool relu, float eps, float monmentum, bool affine, bool track_running_stats) {
    this->num_features = num_features;
    this->relu = relu;
    this->eps = eps;
    this->momentum = monmentum;
    this->affine = affine;
    this->track_running_stats = track_running_stats;

    this->prefix = prefix;

    if(affine) {
        this->weights = new Tensor({num_features}, ONES);
        this->bias = new Tensor({num_features}, ZERO);
    } else {
        // the output is just centralization
        this->weights = nullptr;
        this->bias = nullptr;
    }

    if(this->track_running_stats) {
        this->running_mean = new Tensor({num_features}, ZERO);
        this->running_var = new Tensor({num_features}, ONES);
    } else {
        // use batch statistics(bias estimate)
        this->running_mean = nullptr;
        this->running_var = nullptr;
    }
}

BatchNorm1d::BatchNorm1d(size_t num_features, bool relu, float eps, float monmentum, bool affine, bool track_running_stats) {
    this->num_features = num_features;
    this->relu = relu;
    this->eps = eps;
    this->momentum = monmentum;
    this->affine = affine;
    this->track_running_stats = track_running_stats;

    if(affine) {
        this->weights = new Tensor({num_features}, this->is_training ? ONES : NONE);
        this->bias = new Tensor({num_features}, this->is_training ? ZERO : NONE);
    } else {
        // the output is just centralization
        this->weights = nullptr;
        this->bias = nullptr;
    }

    if(this->track_running_stats) {
        this->running_mean = new Tensor({num_features}, this->is_training ? ZERO : NONE);
        this->running_var = new Tensor({num_features}, this->is_training ? ONES : NONE);
    } else {
        // use batch statistics(bias estimate)
        this->running_mean = nullptr;
        this->running_var = nullptr;
    }
}

BatchNorm1d::~BatchNorm1d() {
    delete running_mean;
    delete running_var;
}

void BatchNorm1d::load_weights() {
    this->weights->fromVec(Configurer::getWeights(this->prefix + "weight"));
    this->bias->fromVec(Configurer::getWeights(this->prefix + "bias"));
    this->running_mean->fromVec(Configurer::getWeights(this->prefix + "running_mean"));
    this->running_var->fromVec(Configurer::getWeights(this->prefix + "running_var"));
}

void BatchNorm1d::load_weights(std::vector<float>& params, const std::string& target) {
    size_t n_data = params.size();
    float* h_data = params.data();
    if(target=="weights") {
        assert(n_data == this->weights->getSize());
        this->weights->load(h_data, n_data);
    } else if(target == "bias") {
        assert(n_data == this->bias->getSize());
        this->bias->load(h_data, n_data);
    } else if(target == "mean") {
        assert(this->running_mean->getSize() == n_data);
        this->running_mean->load(h_data, n_data);
    } else if(target == "var") {
        assert(this->running_var->getSize() == n_data);
        this->running_var->load(h_data, n_data);
    } else {
        ERROR("Load weights %s error!\n", target.c_str());
    }
}

Tensor* BatchNorm1d::forward(Tensor* data) {
    if(!this->track_running_stats){
        // Dependency: Tensor.mean(size_t dim);
        ERROR("not implemented!");
    }
    DimVector shape_o = data->getShape();

    if(this->output == nullptr) {
        this->output = new Tensor(shape_o);
    }
    this->output->reset(shape_o);

    if(this->is_training) {
        this->input = data;
    }

    size_t dim = data->getDim();
    if(dim == 2) {
        DimVector shape = data->getShape();
        int N = shape[0], C = shape[1];

        if(output==nullptr)
            this->output = new Tensor(shape);

        dim3 block(BLOCK_SIZE2D, BLOCK_SIZE2D);
        dim3 grid((C-1)/BLOCK_SIZE2D + 1, (N-1)/BLOCK_SIZE2D+1); 

        kBn1d_l2<<<grid, block>>>(data->getData(), output->getData(), weights->getData(), bias->getData(), running_mean->getData(), running_var->getData(), eps, N, C, this->relu); CHECK_KERNEL();
    } else if(dim == 3) {
        DimVector shape = data->getShape();
        int N = shape[0], C = shape[1], L = shape[2];

        if(output==nullptr)
            this->output = new Tensor(shape);

        dim3 block(BLOCK_SIZE3D, BLOCK_SIZE3D, BATCH_BASE);
        dim3 grid((L-1)/BLOCK_SIZE3D + 1, (C-1)/BLOCK_SIZE3D+1, (N-1)/BATCH_BASE+1); 

        kBn1d_l3<<<grid, block>>>(data->getData(), this->output->getData(), weights->getData(), bias->getData(), running_mean->getData(), running_var->getData(), eps, N, C, L, this->relu); CHECK_KERNEL();
    } else {
        ERROR("Dimension not allowed!");
    }
    return this->output;
}


Tensor* BatchNorm1d::backward(Tensor* gradients){
    return nullptr;
}