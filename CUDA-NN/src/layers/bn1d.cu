#include "hip/hip_runtime.h"
#include "bn1d.cuh"
#include "kernels.cuh"

/* [(N x C) - (C)] / sqrt(C + eps) * (C) + (C) */
__global__ 
void kBn1d_l2(float* d_data, float* d_out, float* weights, float* bias, 
    float* mean, float* var, float eps, int N, int C
) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row >= N || col >= C) return;

    float scaling = rsqrtf(var[col] + eps); // 1 / sqrt(C + eps)
    float norm = (d_data[row * C + col] - mean[col]) * scaling;

    d_out[row * C + col] = norm * weights[col] + bias[col];
}


/* [(N x C x L) - (C)] / sqrt(C + eps) * (C) + (C) */
__global__ 
void kBn1d_l3(float* d_data, float* d_out, float* weights, float* bias, 
    float* mean, float* var, float eps, int N, int C, int L
) {
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if(z >= N || y >= C || x >= L) return;

    float scaling = rsqrtf(var[y] + eps); // 1 / sqrt(C + eps)
    float norm = (d_data[z*C*L + y*L + x] - mean[y]) * scaling;

    d_out[z*C*L + y*L + x] = norm * weights[y] + bias[y];
}

void BatchNorm1d::load_weights() {
    this->weights->fromVec(Configurer::getWeights(this->prefix + "weight"));
    this->bias->fromVec(Configurer::getWeights(this->prefix + "bias"));
    this->running_mean->fromVec(Configurer::getWeights(this->prefix + "running_mean"));
    this->running_var->fromVec(Configurer::getWeights(this->prefix + "running_var"));
}

void BatchNorm1d::load_weights(float *h_weights_data, float *h_bias_data, DimVector weights_shape, DimVector bias_shape) {
    this->weights->initialize(h_weights_data, weights_shape);        
    this->bias->initialize(h_bias_data, bias_shape);
}

void BatchNorm1d::load_weights(float *h_data, DimVector shape, const std::string& target) {
    if(target == "weights") {
        assert(this->weights->getShape() == shape);
        this->weights->initialize(h_data, shape);        
    } else if(target == "bias") {
        assert(this->bias->getShape() == shape);
        this->bias->initialize(h_data, shape);
    } else if(target == "mean") {
        assert(this->running_mean->getShape() == shape);
        this->running_mean->initialize(h_data, shape);
    }else if(target == "var") {
        assert(this->running_var->getShape() == shape);
        this->running_var->initialize(h_data, shape);
    }
}

BatchNorm1d::BatchNorm1d(std::string prefix, size_t num_features, float eps, float monmentum, bool affine, bool track_running_stats) {
    this->num_features = num_features;
    this->eps = eps;
    this->momentum = monmentum;
    this->affine = affine;
    this->track_running_stats = track_running_stats;

    this->prefix = prefix;

    if(affine) {
        this->weights = new Tensor({num_features}, ONES);
        this->bias = new Tensor({num_features}, ZERO);
    } else {
        // the output is just centralization
        this->weights = nullptr;
        this->bias = nullptr;
    }

    if(this->track_running_stats) {
        this->running_mean = new Tensor({num_features}, ZERO);
        this->running_var = new Tensor({num_features}, ONES);
    } else {
        // use batch statistics(bias estimate)
        this->running_mean = nullptr;
        this->running_var = nullptr;
    }
}

BatchNorm1d::~BatchNorm1d() {
    delete running_mean, running_var, weights, bias, input, output, outputBackward;
}

Tensor* BatchNorm1d::forward(Tensor* data) {
    if(!this->track_running_stats){
        // Dependency: Tensor.mean(size_t dim);
        ERROR("not implemented!");
    }

    size_t dim = data->getDim();
    if(dim == 2) {
        DimVector shape = data->getShape();
        int N = shape[0], C = shape[1];
        this->output = new Tensor(shape);

        dim3 block(BLOCK_SIZE2D, BLOCK_SIZE2D);
        dim3 grid((C-1)/BLOCK_SIZE2D + 1, (N-1)/BLOCK_SIZE2D+1); 

        kBn1d_l2<<<grid, block>>>(data->getData(), output->getData(), weights->getData(), bias->getData(), running_mean->getData(), running_var->getData(), eps, N, C); CHECK_KERNEL();
    } else if(dim == 3) {
        DimVector shape = data->getShape();
        int N = shape[0], C = shape[1], L = shape[2];
        this->output = new Tensor(shape);

        dim3 block(BLOCK_SIZE3D, BLOCK_SIZE3D, BLOCK_SIZE3D);
        dim3 grid((L-1)/BLOCK_SIZE3D + 1, (C-1)/BLOCK_SIZE3D+1, (N-1)/BLOCK_SIZE3D+1); 

        kBn1d_l3<<<grid, block>>>(data->getData(), this->output->getData(), weights->getData(), bias->getData(), running_mean->getData(), running_var->getData(), eps, N, C, L); CHECK_KERNEL();
    } else {
        ERROR("Dimension not allowed!");
    }
    return this->output;
}


Tensor* BatchNorm1d::backward(Tensor* gradients){
    return nullptr;
}