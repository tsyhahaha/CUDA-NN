#include "hip/hip_runtime.h"
// 这是程序二的模板程序，我们已经准备好了加载数据集和加载程序一模型参数的部分，请实现CUDA的深度学习推理过程，请严格保持输出格式输出
// nvcc test.cu -o test -Xcompiler "-O3 -std=c++14" -gencode arch=compute_50,code=sm_50 -gencode arch=compute_52,code=sm_52 -gencode arch=compute_53,code=sm_53 -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_62,code=sm_62 -gencode arch=compute_70,code=sm_70 -lhdf5 -lhdf5_cpp

#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <string>
#include <fstream>
#include <map>
#include <dirent.h>
#include <cstring>
#include <H5Cpp.h>

#include "configure.cuh"
#include "layers.cuh"
#include "models.cuh"
#include "tensor.cuh"
#include "utils.cuh"
#include "datasets/dataloader.cuh"

/****************************************************************************************
 * 读取模型参数
 ****************************************************************************************/
// 获取目录中的所有 .txt 文件
std::vector<std::string> get_files_in_directory(const std::string& dir) {
    std::vector<std::string> files;
    DIR* dp;
    struct dirent* entry;
    if ((dp = opendir(dir.c_str())) != NULL) {
        while ((entry = readdir(dp)) != NULL) {
            std::string filename = entry->d_name;
            if (filename.find(".txt") != std::string::npos) {
                files.push_back(filename);
            }
        }
        closedir(dp);
    } else {
        perror("opendir");
    }
    return files;
}

// 读取 .txt 文件并转换为 std::vector<float>
std::vector<float> read_param(const std::string& filepath) {
    std::vector<float> data;
    std::ifstream file(filepath);
    if (file.is_open()) {
        float value;
        while (file >> value) {
            data.push_back(value);
        }
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filepath << std::endl;
    }
    return data;
}

std::map<std::string, std::vector<float>> read_params(std::string dir) {
    // std::string dir = "."; // 当前目录
    std::map<std::string, std::vector<float>> params;

    // 获取目录中的所有 .txt 文件
    std::vector<std::string> param_files = get_files_in_directory(dir);
    for (const auto& file : param_files) {
        std::string filename = file.substr(0, file.find_last_of(".")); // 获取不带扩展名的文件名
        std::cout << filename << "\n";
        params[filename] = read_param(dir + "/" + file);
    }

    // // 访问参数时可以使用 params["conv1_weight"]
    // for (const auto& kv : params) {
    //     std::cout << "Key: " << kv.first << ", Values: ";
    //     // for (const auto& value : kv.second) {
    //     //     std::cout << value << " ";
    //     // }
    //     std::cout << std::endl;
    // }

    return params;
}

/****************************************************************************************
 * 读取训练集数据
 ****************************************************************************************/

using namespace H5;
void read_h5_file(const std::string& file_path, std::vector<std::vector<float>>& list_of_points, std::vector<int>& list_of_labels) {
    try {
        // 打开文件
        H5File file(file_path, H5F_ACC_RDONLY);

        // 获取文件中的所有数据集名称
        std::vector<std::string> dataset_names;
        hsize_t num_objs = file.getNumObjs();
        for (hsize_t i = 0; i < num_objs; i++) {
            dataset_names.push_back(file.getObjnameByIdx(i));
        }

        // 读取每个数据集
        for (const auto& name : dataset_names) {
            DataSet dataset = file.openDataSet(name + "/points");
            DataSpace dataspace = dataset.getSpace();

            // 获取数据集的维度
            hsize_t dims[2];
            dataspace.getSimpleExtentDims(dims, NULL);

            // 读取数据
            std::vector<float> points(dims[0] * dims[1]);
            dataset.read(points.data(), PredType::NATIVE_FLOAT);

            // 存储点云数据
            list_of_points.push_back(points);

            // 读取标签
            Attribute label_attr = file.openGroup(name).openAttribute("label");
            int label;
            label_attr.read(PredType::NATIVE_INT, &label);

            // 存储标签
            list_of_labels.push_back(label);
        }
    } catch (FileIException& error) {
        error.printErrorStack();
    } catch (DataSetIException& error) {
        error.printErrorStack();
    } catch (DataSpaceIException& error) {
        error.printErrorStack();
    } catch (DataTypeIException& error) {
        error.printErrorStack();
    }
}


// 范例kernel函数，无实际作用
__global__ void add_arrays(int* a, int* b, int* c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

int main(int argc, char *argv[]) {
    
    std::string dir = argv[1];  // 第一个参数是程序所在的目录，这个目录是存放前一步训练模型参数文件的目录，从这个目录下读取模型参数文件，相对于这个目录读取测试集点云数据和标签
    // cout << dir;
    
    // 读取模型参数
    auto params = read_params(dir);
    
    Configurer::set_global_weights(params);
    PointNet* pointnet = new PointNet();
    pointnet->load_weights();

    std::string file_path = "/home/tsyhahaha/CUDA-NN/data/splits/test_point_clouds.h5";
    std::vector<std::vector<float>> list_of_points;
    std::vector<int> list_of_labels;
    // 读取训练集数据
    read_h5_file(file_path, list_of_points, list_of_labels);

    DataLoader* dataloader = new DataLoader(list_of_points, list_of_labels, 8, 10240, true);

    // 开始计时，使用chrono计时，不支持其它计时方式
    auto start = std::chrono::high_resolution_clock::now();

    unsigned int right_num = 0;
    unsigned int data_sum = list_of_points.size();
    unsigned int batch_num = dataloader->getBatchNum();
    
    for (size_t i = 0; i < 1; i++) {
        std::vector<int> labels;

        Tensor* input = dataloader->getBatchedData(labels);
        input->transpose(-2, -1);
        Tensor* output = pointnet->forward(input);
        Tensor* pred = output->argmax(-1);
        pred->squeeze();

        float* pred_labels = pred->toHost();

        printM(pred_labels, pred->getShape());
        for(int label: labels) {
            printf("%d ", label);
        }
        printf("\n");
        // clean up
        delete input, output, pred;

    
        // std::cout << "[" << i+1 << "/" << data_sum << "] Points " << i << ", size=" << n_data << ": ";
        // printf("%d", int(pred_label[0]+0.5));
        // // for (const auto& point : list_of_points[i]) {
        // //     std::cout << point << " ";
        // // }
        // std::cout << "\tLabel: " << list_of_labels[i] << std::endl;

        printf("Batch[%d/%d] ", (int)i+1, (int)batch_num);

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> diff = end - start;
        std::cout << "Time: " << diff.count() << "s" << std::endl;
    }
    
    // 向主机端同步以等待所有异步调用的GPU kernel执行完毕，这句必须要有
    hipDeviceSynchronize();

    // 结束计时
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;

    // 输出结果，请严格保持此输出格式，并把0.0001替换成实际的准确率，请不要输出除了此结果之外的任何内容！！！
    std::cout << std::fixed << std::setprecision(4) << diff.count() << right_num << "/" << list_of_points.size() << " (" << (right_num / (float)list_of_points.size()) * 100 << "%)" << std::endl;

    return 0;
}