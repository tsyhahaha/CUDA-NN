#include "hip/hip_runtime.h"
#include "../common.cuh"
#include "configure.cuh"

__global__ void matmul_v2_origin(
    float *d_A, float *d_B, float *d_C, int M, int K, int N
) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * TM;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * TN;
    float tie[TM][TN] = {0.0f};

    if(row >= M || col >= N) return;

    float reg_A[TM] = {0.0f};
    float reg_B[TN] = {0.0f};

    for (int k=0; k < K; k++) {
        for (int i=0; i<TM; i++) {
            if(row + i < M && col < N){
                reg_A[i] = d_A[(row + i) * K + k];
            }
        }
        for(int j=0; j<TN; j++) {
            if(col + j < N && row < M){
                reg_B[j] = d_B[k * N + col + j];
            }
        }

        for (int i=0; i<TM; i++) {
            for(int j=0; j<TN; j++) {
                tie[i][j] += reg_A[i] * reg_B[j];
            }
        }
    }

    for (int i = 0; i < TM; i++){
        for (int j = 0; j< TN; j++){
            if (row + i < M && col + j < N){
                d_C[(row + i) * N + col + j] = tie[i][j];
            }
        }
    }
}

__global__ void matmul_v2_base(
    float *d_A, float *d_B, float *d_C, int M, int K, int N
) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * TM;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * TN;

    int base_row = (blockIdx.y * blockDim.y) * TM + threadIdx.y;
    int base_col = (blockIdx.x * blockDim.x) * TN + threadIdx.x;
    float tie[TM][TN] = {0.0f};

    if(row >= M || col >= N) return;

    // division assumption: BLOCK_SIZE = k * TM/TN
    int stride_row = BLOCK_SIZE;
    int stride_col = BLOCK_SIZE;

    float reg_A[TM] = {0.0f};
    float reg_B[TN] = {0.0f};

    for (int k=0; k < K; k++) {
        for (int i=0; i<TM; i++) {
            if(row + i < M && col < N){
                reg_A[i] = d_A[(row + i) * K + k];
            }
        }

        for(int j=0; j<TN; j++) {
            if(base_col + j * stride_col < N)
                reg_B[j] = d_B[k * N + base_col + j * stride_col];
        }

        for(int j=0; j<TN; j++) {
            for (int i=0; i<TM; i++) {
                tie[i][j] += reg_A[i] * reg_B[j];
            }
        }
    }

    for (int i = 0; i < TM; i++){
        for (int j = 0; j< TN; j++){
            if (row + i < M && base_col + j * stride_col < N){
                d_C[(row + i) * N + base_col + j * stride_col] = tie[i][j];
            }
        }
    }
}

__global__ void matmul_v2(
    float *d_A, float *d_B, float *d_C, int M, int K, int N
) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y)*TM;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*TN;

    if(row >= M || col >= N) return;

    __shared__ float ds_A[BLOCK_SIZE * TM][TILE_SIZE];
    __shared__ float ds_B[TILE_SIZE][BLOCK_SIZE * TN];

    float tie[TM][TN] = {0.0f};
    float reg_A[TM] = {0.0f};
    float reg_B[TN] = {0.0f};

    int phase = (K - 1) / TILE_SIZE + 1;
    for(int p=0; p<phase;p++) {

        for(int i=0; i<TM; i++) {
            if(threadIdx.x < TILE_SIZE) {

                if(row+i < M && p*TILE_SIZE + threadIdx.x < K) {
                    ds_A[threadIdx.y*TM+i][threadIdx.x] = d_A[(row+i) * K + p*TILE_SIZE + threadIdx.x];
                } else {
                    ds_A[threadIdx.y*TM+i][threadIdx.x] = 0.0f;
                }
            }
        }
        

        for(int j=0; j<TN; j++) {
            if(threadIdx.y < TILE_SIZE) {

                if (col + j < N && p*TILE_SIZE + threadIdx.y < K) {
                    ds_B[threadIdx.y][threadIdx.x*TN + j] = d_B[(p*TILE_SIZE + threadIdx.y)*N + col + j];
                } else {
                    ds_B[threadIdx.y][threadIdx.x*TN + j] = 0.0f;
                }
            }
        }
        __syncthreads();

        for (int k=0; k<TILE_SIZE; k++) {
            for (int i=0; i<TM; i++) {
                if(row + i < M) {
                    reg_A[i] = ds_A[threadIdx.y * TM + i][k];
                }
            }

            for(int j=0; j<TN; j++) {
                if(col + j < N) {
                    reg_B[j] = ds_B[k][threadIdx.x * TN + j];
                }
            }
            __syncthreads();

            for(int j=0; j<TN; j++) {
                for (int i=0; i<TM; i++) {
                    tie[i][j] += reg_A[i] * reg_B[j];
                }
            }
        }
    }

    for(int i=0; i<TM; i++) {
        for(int j=0; j<TN; j++) {
            if (row + i < M && col + j < N) {
                d_C[(row + i) * N + col + j] = tie[i][j];
            }
        }
    }
}

int main(){
    // select which kernel to test
    KernelMatmul kernel = matmul_v2;
    const char* name = FUNC_NAME(matmul_v2);

    size_t M = 1024, K = 1024, N = 1024;
    size_t nBytes_A = M * K * sizeof(float);
    size_t nBytes_B = K * N * sizeof(float);
    size_t nBytes_C = M * N * sizeof(float);

    // alloc host memory
    float *h_A, *h_B, *h_C, *h_CC;
    const char* path_A = DATA_PATH;
    const char* path_B = DATA_PATH;
    h_A = loadWeightsFromTxt(path_A, {M, K});
    h_B = loadWeightsFromTxt(path_B, {K, N});
    h_C = (float *)malloc(nBytes_C);
    h_CC = (float *)malloc(nBytes_C); // saved for output of cpu matmul

    // alloc device memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float **)&d_A, nBytes_A));
    CHECK(hipMalloc((float **)&d_B, nBytes_B));
    CHECK(hipMalloc((float **)&d_C, nBytes_C));
    // mv data from Host to Device
    CHECK(hipMemcpy(d_A, h_A, nBytes_A, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes_B, hipMemcpyHostToDevice));

    // kernel launch config
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    int row = (M-1)/(BLOCK_SIZE*TM) + 1, col = (N-1)/(BLOCK_SIZE*TN) + 1; // matmul_v2_base, matmul_v2_origin, matmul_v2_opt1
    dim3 grid(col, row);

    // warm up
    kernel<<<grid, block>>>(d_A, d_B, d_C, M, K, N); CHECK_KERNEL();

    // timer
    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i=0; i<REPEAT_TIMES; i++) {
        kernel<<<grid, block>>>(d_A, d_B, d_C, M, K, N);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);
    printf("Time elapsed on `%s` of %ldx%ld @ %ldx%ld on GPU: %f ms.\n\n", name, M, K, K, N, ker_time/REPEAT_TIMES);

    // get the output of GPU
    CHECK(hipMemcpy(h_C, d_C, nBytes_C, hipMemcpyDeviceToHost));
    CHECK(hipDeviceSynchronize());

    /////////////////////////////////////
    // TODO: check the output, CPU vs GPU
    h_CC = loadWeightsFromTxt("./host_result.txt", {M*N});
    check_result(h_C, h_CC, M*N);
    /////////////////////////////////////

    // free memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_CC);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    hipDeviceReset();
    return 0;
}

